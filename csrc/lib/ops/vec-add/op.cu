#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include <hip/hip_runtime.h>

#include "pmpp/ops/vec_add.hpp"
#include "pmpp/types/cxx_types.hpp"

namespace pmpp::ops
{

__global__ void vecAddKernel(const fp32_t* a, const fp32_t* b, fp32_t* c,
                             int32_t n)
{

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

template <>
void launchVecAdd<fp32_t, Device::CUDA>(const fp32_t* d_A, const fp32_t* d_B,
                                        fp32_t* d_C, size_t n)
{
    uint32_t block_size = 256;
    uint32_t grid_size = (n + block_size - 1) / block_size;

    vecAddKernel<<<grid_size, block_size>>>(d_A, d_B, d_C, int32_t(n));
}

}  // namespace pmpp::ops